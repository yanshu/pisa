#include "mosc3.h"
#include "mosc.h"

#include <stdio.h>

// This improves readability of the equations
#define re (0)
#define im (1)

__device__ void clear_complex_matrix(fType A[][3][2])
{
  // CUDA NOTE: Since I'm on the device, I don't need hipMemset, only
  // memset, because this is called from an execution thread and will
  // refer to the thread's local memory.
  memset(A,0,sizeof(fType)*18);

  // Does this actually work? YES!
  //printf("A: %f %f %f\n",A[0][0][0],A[0][1][0],A[0][2][0]);
  //printf("A: %f %f %f\n",A[1][0][0],A[1][1][0],A[1][2][0]);
  //printf("A: %f %f %f\n",A[2][0][0],A[2][1][0],A[2][2][0]);

}

__device__ void copy_complex_matrix(fType A[][3][2], fType B[][3][2] )
{
  memcpy(B,A,sizeof(fType)*18);
}

__device__ void multiply_complex_matrix(fType A[][3][2], fType B[][3][2], fType C[][3][2] )
{
  for (unsigned i=0; i<3; i++) {
    for (unsigned j=0; j<3; j++) {
      for (unsigned k=0; k<3; k++) {
        C[i][j][0] += A[i][k][re]*B[k][j][re]-A[i][k][im]*B[k][j][im];
        C[i][j][1] += A[i][k][im]*B[k][j][re]+A[i][k][re]*B[k][j][im];
      }
    }
  }
}

__device__ void clear_probabilities(fType Prob[3][3])
{
  memset(Prob,0,sizeof(fType)*9);
}


// Multiply complex 3x3 matrix and 3 vector: W = A X V
__device__ void multiply_complex_matvec( fType A[][3][2], fType V[][2], fType W[][2])
{
  for(unsigned i=0;i<3;i++) {
    W[i][re] = A[i][0][re]*V[0][re]-A[i][0][im]*V[0][im]+
      A[i][1][re]*V[1][re]-A[i][1][im]*V[1][im]+
      A[i][2][re]*V[2][re]-A[i][2][im]*V[2][im] ;
    W[i][im] = A[i][0][re]*V[0][im]+A[i][0][im]*V[0][re]+
      A[i][1][re]*V[1][im]+A[i][1][im]*V[1][re]+
      A[i][2][re]*V[2][im]+A[i][2][im]*V[2][re] ;
  }
}


__device__ void convert_from_mass_eigenstate( int state, int flavor, fType pure[][2],
                                              fType mix[][3][2])
{
  int i,j;
  fType mass[3][2];
  fType conj[3][3][2];
  int    lstate  = state - 1;
  int    factor  = ( flavor > 0 ? -1. : 1. );

  // need the conjugate for neutrinos but not for
  // anti-neutrinos

  for (i=0; i<3; i++) {
    mass[i][0] = ( lstate == i ? 1.0 : 0. );
    mass[i][1] = (                     0. );
  }

  for (i=0; i<3; i++) {
    for (j=0; j<3; j++) {
      conj[i][j][re] =        mix[i][j][re];
      conj[i][j][im] = factor*mix[i][j][im];
    }
  }
  multiply_complex_matvec(conj, mass, pure);

}


__device__ void get_transition_matrix( int nutype, fType Enu, fType rho, fType Len,
                                       fType Aout[][3][2], fType phase_offset,
                                       fType mix[3][3][2], fType dm[3][3])
{

  fType dmMatVac[3][3], dmMatMat[3][3];

  getM(Enu,rho,mix,dm,nutype,dmMatMat,dmMatVac);
  getA(Len,Enu,rho,mix,dmMatVac,dmMatMat,nutype,Aout,phase_offset);

}
