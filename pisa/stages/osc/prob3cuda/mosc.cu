#include "hip/hip_runtime.h"
#include "mosc.h"
#include <stdio.h>

#define elec (0)
#define muon (1)
#define tau  (2)
#define re (0)
#define im (1)

//#define ZERO_CP
static int matrixtype = standard_type;

/* Flag to tell us if we're doing nu_e or nu_sterile matter effects */
//static NuType matterFlavor = nue_type;


/***********************************************************************
  getM
  Compute the matter-mass vector M, dM = M_i-M_j and
  and dMimj. type<0 means anti-neutrinos type>0 means "real" neutrinos
***********************************************************************/
__device__ void getM(fType Enu, fType rho,
                     fType Mix[][3][2], fType dmVacVac[][3], int antitype,
                     fType dmMatMat[][3], fType dmMatVac[][3])
{
  int i, j, k;
  fType alpha, beta, gamma, fac=0.0, arg, tmp;
  fType alphaV, betaV, gammaV, argV, tmpV;
  fType theta0, theta1, theta2;
  fType theta0V, theta1V, theta2V;
  fType mMatU[3], mMatV[3], mMat[3];
  fType tworttwoGf = 1.52588e-4;

  /* Equations (22) fro Barger et.al.*/
  /* Reverse the sign of the potential depending on neutrino type */
  //if (matterFlavor == nue_type) {
  /* If we're doing matter effects for electron neutrinos */
  if (antitype<0) fac =  tworttwoGf*Enu*rho; /* Anti-neutrinos */
  else        fac = -tworttwoGf*Enu*rho; /* Real-neutrinos */
  //}
  //else if (matterFlavor == sterile_type) {
  /* If we're doing matter effects for sterile neutrinos */
  //if (antitype<0) fac = -0.5*tworttwoGf*Enu*rho; /* Anti-neutrinos */

  //   else        fac =  0.5*tworttwoGf*Enu*rho; /* Real-neutrinos */
  // }
  /* The strategy to sort out the three roots is to compute the vacuum
   * mass the same way as the "matter" masses are computed then to sort
   * the results according to the input vacuum masses
   */

  alpha  = fac + dmVacVac[0][1] + dmVacVac[0][2];
  alphaV = dmVacVac[0][1] + dmVacVac[0][2];

#ifndef ZERO_CP
  beta = dmVacVac[0][1]*dmVacVac[0][2] +
    fac*(dmVacVac[0][1]*(1.0 - Mix[elec][1][re]*Mix[elec][1][re] -
                         Mix[elec][1][im]*Mix[elec][1][im]) +
         dmVacVac[0][2]*(1.0 - Mix[elec][2][re]*Mix[elec][2][re] -
                         Mix[elec][2][im]*Mix[elec][2][im]));
  betaV = dmVacVac[0][1]*dmVacVac[0][2];

#else
  beta = dmVacVac[0][1]*dmVacVac[0][2] +
    fac*(dmVacVac[0][1]*(1.0 - Mix[elec][1][re]*Mix[elec][1][re]) +
         dmVacVac[0][2]*(1.0- Mix[elec][2][re]*Mix[elec][2][re]));
  betaV = dmVacVac[0][1]*dmVacVac[0][2];
#endif

#ifndef ZERO_CP
  gamma = fac*dmVacVac[0][1]*dmVacVac[0][2]*
    (Mix[elec][0][re]*Mix[elec][0][re]+Mix[elec][0][im]*Mix[elec][0][im]);
  gammaV = 0.0;
#else
  gamma = fac*dmVacVac[0][1]*dmVacVac[0][2]*
    (Mix[elec][0][re]*Mix[elec][0][re]);
  gammaV = 0.0;
#endif

  /* Compute the argument of the arc-cosine */
  tmp = alpha*alpha-3.0*beta;
  tmpV = alphaV*alphaV-3.0*betaV;
  if (tmp<0.0) {
    // fprintf(stderr, "getM: alpha^2-3*beta < 0 !\n");
    tmp = 0.0;
  }

  /* Equation (21) */
  arg = (2.0*alpha*alpha*alpha-9.0*alpha*beta+27.0*gamma)/
    (2.0*sqrt(tmp*tmp*tmp));
  if (fabs(arg)>1.0) arg = arg/fabs(arg);
  argV = (2.0*alphaV*alphaV*alphaV-9.0*alphaV*betaV+27.0*gammaV)/
    (2.0*sqrt(tmpV*tmpV*tmpV));
  if (fabs(argV)>1.0) argV = argV/fabs(argV);

  /* These are the three roots the paper refers to */
  theta0 = acos(arg)/3.0;
  theta1 = theta0-(2.0*M_PI/3.0);
  theta2 = theta0+(2.0*M_PI/3.0);
  theta0V = acos(argV)/3.0;
  theta1V = theta0V-(2.0*M_PI/3.0);
  theta2V = theta0V+(2.0*M_PI/3.0);

  mMatU[0] = mMatU[1] = mMatU[2] = -(2.0/3.0)*sqrt(tmp);
  mMatU[0] *= cos(theta0); mMatU[1] *= cos(theta1); mMatU[2] *= cos(theta2);

  tmp = dmVacVac[0][0] - alpha/3.0;
  mMatU[0] += tmp; mMatU[1] += tmp; mMatU[2] += tmp;
  mMatV[0] = mMatV[1] = mMatV[2] = -(2.0/3.0)*sqrt(tmpV);
  mMatV[0] *= cos(theta0V); mMatV[1] *= cos(theta1V); mMatV[2] *= cos(theta2V);
  tmpV = dmVacVac[0][0] - alphaV/3.0;

  mMatV[0] += tmpV; mMatV[1] += tmpV; mMatV[2] += tmpV;

  /* Sort according to which reproduce the vaccum eigenstates */
  for (i=0; i<3; i++) {
    tmpV = fabs(dmVacVac[i][0]-mMatV[0]);
    k = 0;
    for (j=1; j<3; j++) {
      tmp = fabs(dmVacVac[i][0]-mMatV[j]);
      if (tmp<tmpV) {
        k = j;
        tmpV = tmp;
      }
    }
    mMat[i] = mMatU[k];
  }

  for (i=0; i<3; i++) {
    for (j=0; j<3; j++) {
      dmMatMat[i][j] = mMat[i] - mMat[j];
      dmMatVac[i][j] = mMat[i] - dmVacVac[j][0];
    }
 }
}

/***********************************************************************
 getA
 Calculate the transition amplitude matrix A (equation 10)
***********************************************************************/
__device__ void getA(fType L, fType E, fType rho,
                     fType Mix[][3][2], fType dmMatVac[][3],
                     fType dmMatMat[][3], int antitype, fType A[3][3][2],
                     fType phase_offset)
{

  /*
    DARN - looks like this is all junk...more debugging needed...
  */

  //int n, m, i, j, k;
  fType /*fac=0.0,*/ arg, c, s;
  // TCA ADDITION: set equal to 0!
  fType X[3][3][2] = {0.0};
  fType product[3][3][3][2] = {0.0};
  /* (1/2)*(1/(h_bar*c)) in units of GeV/(eV^2-km) */
  const fType LoEfac = 2.534;

  if ( phase_offset==0.0 )
    {
      get_product(L, E, rho, Mix, dmMatVac, dmMatMat, antitype, product);
    }

  /////////////// product is JUNK /////////////

  for (int i=0; i<3; i++){
    for (int j=0; j<3; j++) {
  //printf(" product[%d][%d]: %f, %f\n",i,j,*product[i][j][0],*product[i][j][1]);
  //printf(" A[%d][%d]: %f, %f\n",i,j,A[i][j][0],A[i][j][1]);
    }
  }

  /* Make the sum with the exponential factor */
  //hipMemset(X, 0, 3*3*2*sizeof(fType));
  //memset(X, 0, 3*3*2*sizeof(fType));
  for (int k=0; k<3; k++)
    {
      arg = -LoEfac*dmMatVac[k][0]*L/E;
      if ( k==2 ) arg += phase_offset ;
      c = cos(arg);
      s = sin(arg);
      for (int i=0; i<3; i++)
        {
          for (int j=0; j<3; j++)
            {
#ifndef ZERO_CP
              X[i][j][re] += c*product[i][j][k][re] - s*product[i][j][k][im];
              X[i][j][im] += c*product[i][j][k][im] + s*product[i][j][k][re];
#else
              X[i][j][re] += c*product[i][j][k][re];
              X[i][j][im] += s*product[i][j][k][re];
#endif
            }
        }
    }


  /* Compute the product with the mixing matrices */
  for(int i=0; i < 3; i++)
    for(int j = 0; j < 3; j++)
      for(int k = 0; k < 2; k++)
        A[i][j][k] = 0;

  for (int n=0; n<3; n++) {
    for (int m=0; m<3; m++) {
      for (int i=0; i<3; i++) {
        for (int j=0; j<3; j++) {
#ifndef ZERO_CP
          A[n][m][re] +=
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][re] +
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][im] +
            Mix[n][i][im]*X[i][j][re]*Mix[m][j][im] -
            Mix[n][i][im]*X[i][j][im]*Mix[m][j][re];
          //printf("\nregret %f %f %f",Mix[n][i][re], X[i][j][im], Mix[m][j][im]);
          A[n][m][im] +=
            Mix[n][i][im]*X[i][j][im]*Mix[m][j][im] +
            Mix[n][i][im]*X[i][j][re]*Mix[m][j][re] +
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][re] -
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][im];
#else
          A[n][m][re] +=
            Mix[n][i][re]*X[i][j][re]*Mix[m][j][re];
          A[n][m][im] +=
            Mix[n][i][re]*X[i][j][im]*Mix[m][j][re];
#endif
          //printf("\n %i %i %i A %f", n, m, re, A[n][m][re]);
        }
      }
    }
  }

  //printf("(getA) Aout: %f\n",A[0][0][0]);

}


__device__ void get_product(fType L, fType E, fType rho,fType Mix[][3][2],
                            fType dmMatVac[][3], fType dmMatMat[][3],
                            int antitype,
                            fType product[][3][3][2])
{

  fType fac=0.0;
  fType twoEHmM[3][3][3][2];
  fType tworttwoGf = 1.52588e-4;

  /* (1/2)*(1/(h_bar*c)) in units of GeV/(eV^2-km) */
  /* Reverse the sign of the potential depending on neutrino type */
  //if (matterFlavor == nue_type) {

  /* If we're doing matter effects for electron neutrinos */
  if (antitype<0) fac =  tworttwoGf*E*rho; /* Anti-neutrinos */
  else        fac = -tworttwoGf*E*rho; /* Real-neutrinos */
  //  }

  /*
      else if (matterFlavor == sterile_type) {
      // If we're doing matter effects for sterile neutrinos
      if (antitype<0) fac = -0.5*tworttwoGf*E*rho; // Anti-neutrinos
      else        fac =  0.5*tworttwoGf*E*rho; // Real-neutrinos
      } */

  /* Calculate the matrix 2EH-M_j */
  for (int n=0; n<3; n++) {
    for (int m=0; m<3; m++) {

#ifndef ZERO_CP
      twoEHmM[n][m][0][re] =
        -fac*(Mix[0][n][re]*Mix[0][m][re]+Mix[0][n][im]*Mix[0][m][im]);
      twoEHmM[n][m][0][im] =
        -fac*(Mix[0][n][re]*Mix[0][m][im]-Mix[0][n][im]*Mix[0][m][re]);

      twoEHmM[n][m][1][re] = twoEHmM[n][m][2][re] = twoEHmM[n][m][0][re];
      twoEHmM[n][m][1][im] = twoEHmM[n][m][2][im] = twoEHmM[n][m][0][im];

#else

      twoEHmM[n][m][0][re] =
        -fac*(Mix[0][n][re]*Mix[0][m][re]);
      twoEHmM[n][m][0][im] = 0 ;
      twoEHmM[n][m][1][re] = twoEHmM[n][m][2][re] = twoEHmM[n][m][0][re];
      twoEHmM[n][m][1][im] = twoEHmM[n][m][2][im] = twoEHmM[n][m][0][im];

#endif

      if (n==m) for (int j=0; j<3; j++)
                  twoEHmM[n][m][j][re] -= dmMatVac[j][n];
    }
  }

  /* Calculate the product in eq.(10) of twoEHmM for j!=k */
  for (int i=0; i<3; i++) {
    for (int j=0; j<3; j++) {
      for (int k=0; k<3; k++) {

#ifndef ZERO_CP

        product[i][j][0][re] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][re] -
          twoEHmM[i][k][1][im]*twoEHmM[k][j][2][im];
        product[i][j][0][im] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][im] +
          twoEHmM[i][k][1][im]*twoEHmM[k][j][2][re];
        product[i][j][1][re] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][re] -
          twoEHmM[i][k][2][im]*twoEHmM[k][j][0][im];
        product[i][j][1][im] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][im] +
          twoEHmM[i][k][2][im]*twoEHmM[k][j][0][re];
        product[i][j][2][re] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][re] -
          twoEHmM[i][k][0][im]*twoEHmM[k][j][1][im];
        product[i][j][2][im] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][im] +
          twoEHmM[i][k][0][im]*twoEHmM[k][j][1][re];

#else
        product[i][j][0][re] +=
          twoEHmM[i][k][1][re]*twoEHmM[k][j][2][re];
        product[i][j][1][re] +=
          twoEHmM[i][k][2][re]*twoEHmM[k][j][0][re];
        product[i][j][2][re] +=
          twoEHmM[i][k][0][re]*twoEHmM[k][j][1][re];

#endif
      }
#ifndef ZERO_CP

      product[i][j][0][re] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][0][im] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][1][re] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][1][im] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][2][re] /= (dmMatMat[2][0]*dmMatMat[2][1]);
      product[i][j][2][im] /= (dmMatMat[2][0]*dmMatMat[2][1]);

#else
      product[i][j][0][re] /= (dmMatMat[0][1]*dmMatMat[0][2]);
      product[i][j][1][re] /= (dmMatMat[1][2]*dmMatMat[1][0]);
      product[i][j][2][re] /= (dmMatMat[2][0]*dmMatMat[2][1]);

#endif
    }
  }
}
